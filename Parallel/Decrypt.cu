#include "hip/hip_runtime.h"
#include "ppmHelper.h"
#include "mp3Helper.h"

#include <bits/stdc++.h>

using namespace std;

#define THREADS_PER_BLOCK 512

__global__
void decrypt(unsigned char * inputImageData, int width, int height,
        char * audioData, int audioSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < audioSize){
        unsigned char audioByte = 0;

        audioByte |= (inputImageData[index * 8 + 0] & 1) << 7;
        audioByte |= (inputImageData[index * 8 + 1] & 1) << 6;
        audioByte |= (inputImageData[index * 8 + 2] & 1) << 5;
        audioByte |= (inputImageData[index * 8 + 3] & 1) << 4;
        audioByte |= (inputImageData[index * 8 + 4] & 1) << 3;
        audioByte |= (inputImageData[index * 8 + 5] & 1) << 2;
        audioByte |= (inputImageData[index * 8 + 6] & 1) << 1;
        audioByte |= (inputImageData[index * 8 + 7] & 1) << 0;
        
        audioData[index] = audioByte;
    }
}


int main(int argc, char *argv[]){

    char *inputImageFile = argv[1];
    int audioSize = atoi(argv[2]);

    // Read input image
    PPMimg *inpImg = readPPM(inputImageFile);
    int width = inpImg->width;
    int height = inpImg->height;
    int totPixels = width * height;

    PPMpixel *inData = inpImg->data;
    PPMpixel *outData = (PPMpixel *)malloc(sizeof(PPMpixel) * totPixels);
    unsigned char *inputImageData = ppmTochar(inData, width, height);

    // Copy input image to device memory
    unsigned char *d_inputImageData;
    hipMalloc((void**)&d_inputImageData, totPixels * 3ll);

    hipMemcpy(d_inputImageData,inputImageData,totPixels * 3ll,hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------//

    // Memory allocation for extracted audio
    char *extractedAudioData = (char *)malloc(audioSize);
    char *d_extractedAudioData;
    hipMalloc((void**)&d_extractedAudioData, audioSize);
    //--------------------------------------------------------------------------//

    // Invoke Kernel
    dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    dim3 gridDim((audioSize-1)/THREADS_PER_BLOCK + 1, 1, 1);

    decrypt<<<blockDim, gridDim>>>(d_inputImageData, width, height, d_extractedAudioData, audioSize);
    //--------------------------------------------------------------------------//

    // Copying result to host
    hipMemcpy(extractedAudioData, d_extractedAudioData, audioSize ,hipMemcpyDeviceToHost);


    // Writing back audio file
    char outputAudioFile[] = "././Dataset/parallel_output.mp3";
    writeMP3(outputAudioFile, extractedAudioData, audioSize);
    //--------------------------------------------------------------------------//
}