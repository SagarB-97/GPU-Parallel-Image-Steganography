#include "hip/hip_runtime.h"
#include "ppmHelper.h"
#include "mp3Helper.h"

#include <bits/stdc++.h>

using namespace std;

#define THREADS_PER_BLOCK 512

__global__
void encrypt(unsigned char * inputImageData, unsigned char * outputImageData, int width, int height,
        char * audioData, int audioSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < audioSize){
        unsigned char audioByte = (unsigned char)audioData[index];

        outputImageData[index * 8 + 0] = (inputImageData[index * 8 + 0] | 1) & (254 + ((audioByte>>7) & 1));
        outputImageData[index * 8 + 1] = (inputImageData[index * 8 + 1] | 1) & (254 + ((audioByte>>6) & 1));
        outputImageData[index * 8 + 2] = (inputImageData[index * 8 + 2] | 1) & (254 + ((audioByte>>5) & 1));
        outputImageData[index * 8 + 3] = (inputImageData[index * 8 + 3] | 1) & (254 + ((audioByte>>4) & 1));
        outputImageData[index * 8 + 4] = (inputImageData[index * 8 + 4] | 1) & (254 + ((audioByte>>3) & 1));
        outputImageData[index * 8 + 5] = (inputImageData[index * 8 + 5] | 1) & (254 + ((audioByte>>2) & 1));
        outputImageData[index * 8 + 6] = (inputImageData[index * 8 + 6] | 1) & (254 + ((audioByte>>1) & 1));
        outputImageData[index * 8 + 7] = (inputImageData[index * 8 + 7] | 1) & (254 + ((audioByte>>0) & 1));
    }
}


int main(int argc, char *argv[]){

    char *inputImageFile = argv[1];
    char *inputAudioFile = argv[2];

    // Read input image
    PPMimg *inpImg = readPPM(inputImageFile);
    int width = inpImg->width;
    int height = inpImg->height;
    int totPixels = width * height;

    PPMpixel *inData = inpImg->data;
    PPMpixel *outData = (PPMpixel *)malloc(sizeof(PPMpixel) * totPixels);
    unsigned char * inputImageData = ppmTochar(inData, width, height);
    unsigned char * outputImageData = (unsigned char *)malloc(totPixels * 3ll);

    // Copy input image to device memory
    unsigned char *d_inputImageData, *d_outputImageData;
    hipMalloc((void**)&d_inputImageData, totPixels * 3ll);
    hipMalloc((void**)&d_outputImageData, totPixels * 3ll);

    hipMemcpy(d_inputImageData,inputImageData,totPixels * 3ll,hipMemcpyHostToDevice);
    hipMemcpy(d_outputImageData,inputImageData,totPixels * 3ll,hipMemcpyHostToDevice);    
    //--------------------------------------------------------------------------//


    // Read input audio file
    MP3File *inpAudio = readMP3(inputAudioFile);
    char *audioData = inpAudio->data;

    // Copy audio file to device memory
    char *d_audioData;
    hipMalloc((void**)&d_audioData, inpAudio->size);
    hipMemcpy(d_audioData, audioData, inpAudio->size, hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------//

    cout << "Size of audio file = " << inpAudio->size << " bytes ("
         << (inpAudio->size * 8) << " bits)\n";
    cout << "Size of image file = " << totPixels * 3 << " bytes\n";

    // Invoke Kernel
    int audioSize = inpAudio -> size;
    dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    dim3 gridDim((audioSize-1)/THREADS_PER_BLOCK + 1, 1, 1);

    encrypt<<<blockDim, gridDim>>>(d_inputImageData, d_outputImageData, width, height, d_audioData, 
            audioSize);
    //--------------------------------------------------------------------------//

    // Writing result to host
    hipMemcpy(outputImageData, d_outputImageData, totPixels * 3 ,hipMemcpyDeviceToHost);

    // Writing back output image
    char outputImageFile[] = "././Dataset/parallel_output.ppm";
    writePPM(outputImageFile, outputImageData, inpImg->width, inpImg->height, 3);
    //--------------------------------------------------------------------------//

    // Free memory
    free(audioData);
    hipFree(d_inputImageData);
    hipFree(d_outputImageData);
    hipFree(d_audioData);
}